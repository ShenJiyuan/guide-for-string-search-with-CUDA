#include "hip/hip_runtime.h"

#include "hip/hip_runtime.h"
#include ""

#include <stdio.h>
#include <iostream>
#include <hip/device_functions.h>

#define BlockNum 65535
#define ThreadNum 512

//=====================================================kmp�㷨Ԥ����====================================================//
void getNext(char *pattern, int pattern_len, int *next)
{
	int len = 0;  // ��¼ǰһ��[�ƥ���ǰ׺�ͺ�׺]�ĳ���
	int i;
	next[0] = 0; // next[0] ������ 0
	i = 1;
	// the loop calculates next[i] for i = 1 to pattern_len-1
	while (i < pattern_len)
	{
		if (pattern[i] == pattern[len])
		{
			len++;
			next[i] = len;
			i++;
		}
		else // (pat[i] != pat[len])
		{
			if (len == 0)
			{
				next[i] = len; //û��һ��ƥ���
				i++;
			}
			else // ��� (len == 0)
			{
				// ����ط�������. ����������� AAACAAAA ,i = 7.
				len = next[len - 1];
				// ����, ע�� i ������ط���û������
			}
		}
	}
}
//======================================================kmp�㷨ʵ��=====================================================//
__device__ void KMP(char *pattern, int pattern_len, char *array, int array_len, int *answer, int *next, int cursor, int end)
{
	//ÿ���̴߳���һ��pattern_len������index�Ĳ���Ϊid*pattern_len
	int j = 0;//j as index for pattern
	//cursor as index for array
	while (cursor < end)
	{
		if (pattern[j] == array[cursor])
		{
			j++;
			cursor++;
		}
		if (j == pattern_len)
		{
			//printf("Found pattern at index %d \n", i - j);
			answer[cursor - j] = 1;
			j = next[j - 1];
		}
		// mismatch after j matches
		else if (pattern[j] != array[cursor])
		{
			// Do not match next[0..next[j-1]] characters,
			// they will match anyway
			if (j != 0)
				j = next[j - 1];
			else
				cursor = cursor + 1;
		}
	}
}
//=====================================================kernel��������===================================================//
__global__ void kmp_kernel(char *arrayIn, char *patternIn, int *answerIn, int *next, int array_len, int pattern_len)
{
	int id = blockIdx.x*blockDim.x + threadIdx.x;
	int offset= 2 * pattern_len; 
	int cursor, end;

		if (id < 0.5*(array_len / pattern_len))
		{
			cursor = id*offset;
			end = id*offset + offset;
		}
		else
		{ //�����߳�
			cursor = (id % ((array_len / pattern_len) / 2))*offset + offset - pattern_len;
			end = (id % ((array_len / pattern_len) / 2))*offset + offset + pattern_len;
		}

	KMP(patternIn, pattern_len, arrayIn, array_len, answerIn, next, cursor, end);
	//����������������� global memory �� shared memory
	//__shared__ char array[blockDim.x+2*pattern_len];
}
//========================================================main����======================================================//
int main()
{
	//=========================================================================//
	//error handling
	hipError_t r;
	//host copies declaration
	char *array, *pattern; int *answer;
	//device copies declaration
	char *d_array, *d_pattern; int *d_answer;

	//=========================================================================//
	//input file operations & host arrays
	FILE * infile = fopen("input.txt", "r");
	if (infile == NULL){
		printf("ERROR:Could not open file '%s'.\n", "infile");
		exit(-1);
	}
	char readTemp;
	int array_len = 0; int pattern_len = 0;
	while ((readTemp = fgetc(infile)) != '\n') array_len++;
	while ((readTemp = fgetc(infile)) != '\n') pattern_len++;

	if (pattern_len > array_len || pattern_len < 0 || array_len < 0) { printf("ERROR INPUT!"); return 0; }
	bool zero_flag = false;
	if (pattern_len == 0 && array_len == 0) zero_flag = true;

	fseek(infile, 0, SEEK_SET);
	array = (char*)malloc(array_len*sizeof(char));
	pattern = (char*)malloc(pattern_len*sizeof(char));
	answer = (int*)malloc(array_len*sizeof(int));
	int readTemp1 = 0;
	while ((readTemp = fgetc(infile)) != '\n') { array[readTemp1] = readTemp; readTemp1++; }
	readTemp1 = 0;
	while ((readTemp = fgetc(infile)) != '\n') { pattern[readTemp1] = readTemp; readTemp1++; }
	fclose(infile);
	for (readTemp1 = 0; readTemp1 < array_len; readTemp1++) answer[readTemp1] = 0;

	//=========================================================================//
	//device arrays allocation
	r = hipMalloc((void**)&d_array, sizeof(char)*array_len);
	printf("hipMalloc d_array : %s\n", hipGetErrorString(r));
	r = hipMalloc((void**)&d_pattern, sizeof(char)*pattern_len);
	printf("hipMalloc d_pattern : %s\n", hipGetErrorString(r));
	r = hipMalloc((void**)&d_answer, sizeof(int)*array_len);
	printf("hipMalloc d_answer : %s\n", hipGetErrorString(r));

	int* r_next = (int*)malloc(pattern_len*sizeof(int));
	//device
	int* next;
	r = hipMalloc((void**)&next, sizeof(int)*pattern_len);
	printf("hipMalloc next : %s\n", hipGetErrorString(r));
	//preprocessing
	getNext(pattern, pattern_len, r_next);

	//=========================================================================//
	//hipMemcpy for parameters
	r = hipMemcpy(d_array, array, sizeof(char)*array_len, hipMemcpyHostToDevice);
	printf("Memory copy H->D d_array : %s\n", hipGetErrorString(r));
	r = hipMemcpy(d_pattern, pattern, sizeof(char)*pattern_len, hipMemcpyHostToDevice);
	printf("Memory copy H->D d_pattern : %s\n", hipGetErrorString(r));
	//copy for next
	r = hipMemcpy(next, r_next, sizeof(int)*pattern_len, hipMemcpyHostToDevice);
	printf("Memory copy H->D d_pattern : %s\n", hipGetErrorString(r));

	//=========================================================================//
		int threads = (array_len / pattern_len);//ÿ���̴߳���һ��pattern���ȵ��ַ���
		int blocks = (threads / 1024) + 1;
		//call kernel
		kmp_kernel << <blocks, 2014 >> >(d_array, d_pattern, d_answer, next, array_len, pattern_len);

	r = hipDeviceSynchronize();
	printf("Device synchronize : %s\n", hipGetErrorString(r));
	//=========================================================================//
	//hipMemcpy for result
	r = hipMemcpy(answer, d_answer, sizeof(int)*array_len, hipMemcpyDeviceToHost);
	printf("Memory copy D->H answer : %s\n", hipGetErrorString(r));

	//test
	//int test;
	//for (test = 0; test < array_len; test++) printf("pos[%d]=%d\n", test, answer[test]);

	//=========================================================================//
	//output file operations
	FILE * outfile = fopen("output.txt", "w+");
	if (outfile == NULL){
		printf("ERROR:Could not open file '%s'.\n", "outfile");
		exit(-1);
	}
	if (zero_flag == false)
	{
		int writeTemp;
		bool flag = 0;
		for (writeTemp = 0; writeTemp < array_len; writeTemp++)
		if (answer[writeTemp] == 1)
		{
			if (flag == 0) flag = 1;
			fprintf(outfile, "%d\n", writeTemp);
		}
		if (flag == 0) fprintf(outfile, "Not found.");
	}
	else fprintf(outfile,"Null input.");
	fclose(outfile);

	//=========================================================================//
	//pointers free (host&device)
	free(array); free(pattern); free(answer); free(r_next);
	hipFree(d_array); hipFree(d_pattern); hipFree(d_answer); hipFree(next);

	return 1;
}
//======================================================================================================================//